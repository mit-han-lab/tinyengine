#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <assert.h>
#include "matmul.h"
#include <sys/time.h>
#include <stdlib.h>
#include <cstdlib>
#include <iostream>

const int threadDim = 32;
const int TILE_SIZE = threadDim;
__global__ void matrixMul_blockC(float *A, float *B, float *C, int A_row, int A_column, int B_column){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	float acc = 0;
	for (int k = 0; k < A_column; k++)
		acc += A[j * A_column + k] * B[k * B_column + i];
	C[j * B_column +i] = acc;
}

__global__ void matrixMultiplyShared(const float *A, const float *B, float *C, int A_row, int A_column, int B_column) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float As[TILE_SIZE][TILE_SIZE];
	__shared__ float Bs[TILE_SIZE][TILE_SIZE];

	float value = 0;

	for (int i = 0; i < A_column / TILE_SIZE; i++){
		As[threadIdx.y][threadIdx.x] = A[(blockIdx.y * TILE_SIZE + threadIdx.y) * A_column + TILE_SIZE * i + threadIdx.x];
		Bs[threadIdx.y][threadIdx.x] = B[(i * TILE_SIZE + threadIdx.y) * B_column + blockIdx.x * TILE_SIZE + threadIdx.x];

		__syncthreads();

		for (int k = 0; k < TILE_SIZE; k++)
			value += As[threadIdx.y][k] * Bs[k][threadIdx.x];

		__syncthreads();
	}


	C[row * B_column + col] = value;
}

namespace matmul{

	void MatmulOperator::mat_mul_cuda(const struct matmul_params *params){
		const struct matrix *A = &params->A, *B = &params->B, *C = &params->C;
		assert(A->column == B->row);
		assert(C->column == B->column);
		assert(C->row == A->row);

		float *d_A;
		float *d_B;
		float *d_C;

		// Initailize C
		/*for (int i = 0; i < C->row; i++)
		  for (int j = 0; j < C->column; j++)
		  C->data_ptr[j + C->column * i] = 0;*/

		// Allocate memory
		hipMalloc(&d_A, A->column*A->row*sizeof(float));
		hipMalloc(&d_B, B->column*B->row*sizeof(float));
		hipMalloc(&d_C, C->column*C->row*sizeof(float));

		// Copy data to GPU
		hipMemcpy(d_A, A->data_ptr, A->column*A->row*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_B, B->data_ptr, B->column*B->row*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_C, C->data_ptr, C->column*C->row*sizeof(float), hipMemcpyHostToDevice);

		// Make sure we can break the input matrix into blocks
		assert(A->column % threadDim == 0);
		assert(A->row % threadDim == 0);
		assert(B->column % threadDim == 0);
		const dim3 threadsPerBlock(threadDim, threadDim);
		const dim3 numBlocks(C->column / threadsPerBlock.x, C->row / threadsPerBlock.y);

		// Invoke the cuda imp.

		struct timeval start, end;
		gettimeofday(&start, NULL);
		//matrixMul_blockC<<< numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, A->row, A->column, B->column);
		matrixMultiplyShared<<< numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, A->row, A->column, B->column);
		hipDeviceSynchronize();
		gettimeofday(&end, NULL);
		int us = interval_to_us(&start, &end);
		std::cout << "cuda kernel: " << us / 1000 << " ms" << std::endl;

		// Get the result back
		hipMemcpy(C->data_ptr, d_C, C->column*C->row*sizeof(float), hipMemcpyDeviceToHost);
	}
}
